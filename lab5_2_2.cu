/* Command to compile on Windows:
nvcc .\lab5_2_2.cu -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\BuildTools\VC\Tools\MSVC\14.29.30133\bin\Hostx64\x64"

Output should be:
a: [22, 13, 16, 5]
b: [5, 22, 17, 37]
c: [27, 35, 33, 42]
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(int *c, int *a, int *b) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void array2str(char *str, int *array, int n, int str_size) {
    int written = 0;
    written += snprintf(str + written, str_size - written, "[");
    for (int idx = 0; idx < n - 1; idx++) {
        written += snprintf(str + written, str_size - written, "%i, ", *(array + idx));
    }
    written += snprintf(str + written, str_size - written, "%i]", *(array + n - 1));
    return;
}

int main(void) {
    /* Intiialize inputs (CPU) */
    const int N = 4;
    int a[N] = {22, 13, 16, 5};
    int b[N] = {5, 22, 17, 37};
    int c[N];
    char str_a[80];
    char str_b[80];
    array2str(str_a, a, N, 80);
    array2str(str_b, b, N, 80);
    printf("a: %s\n", str_a);
    printf("b: %s\n", str_b);

    /* Allocate memory for calculation on GPU */
    int *gpu_a;
    int *gpu_b;
    int *gpu_c;
    hipMalloc((void**) &gpu_a, sizeof(int) * N);
    hipMalloc((void**) &gpu_b, sizeof(int) * N);
    hipMalloc((void**) &gpu_c, sizeof(int) * N);

    /* Copy inputs to GPU */
    hipMemcpy(gpu_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu_b, b, sizeof(int) * N, hipMemcpyHostToDevice);
 
    /* Do the thing */
    vector_add<<<1, N>>>(gpu_c, gpu_a, gpu_b);
    hipMemcpy(c, gpu_c, sizeof(int) * N, hipMemcpyDeviceToHost);

    /* Remember to clean up after ourselves */
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(gpu_c);

    /* Print result */
    char str_c[80];
    array2str(str_c, c, N, 80);
    printf("c: %s\n", str_c);

    return 0;
}